#include "hip/hip_runtime.h"
#include "cast.cuh"
#include "cublas.cuh"
#include "gemm_simple.cuh"
#include "rng.cuh"

void test(size_t m, size_t n, size_t k, size_t repeat) {
    half *c_cublas, *c_our, *a, *b;
    CUDA_CALL(hipMalloc(&c_cublas, m * n * sizeof(half)));
    CUDA_CALL(hipMalloc(&c_our, m * n * sizeof(half)));
    CUDA_CALL(hipMalloc(&a, m * k * sizeof(half)));
    CUDA_CALL(hipMalloc(&b, k * n * sizeof(half)));

    RNG gen(1234ULL);
    float *rand;
    CUDA_CALL(hipMalloc(&rand, std::max(m * k, k * n) * sizeof(float)));
    gen.rand(rand, m * k);
    cast(a, rand, m * k);
    gen.rand(rand, k * n);
    cast(b, rand, k * n);

    hipStream_t stream;
    CUDA_CALL(hipStreamCreate(&stream));

    hipEvent_t start, end;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&end));

    float cublas_time = 0.0;
    Cublas cublas(stream);
    for (size_t i = 0; i < repeat; i++) {
        CUDA_CALL(hipEventRecord(start, stream));
        cublas.gemm(c_cublas, a, b, m, n, k);
        CUDA_CALL(hipEventRecord(end, stream));

        CUDA_CALL(hipEventSynchronize(end));
        float ms;
        CUDA_CALL(hipEventElapsedTime(&ms, start, end));
        cublas_time += ms;
    }
    printf("cublas time: %.3f ms\n", cublas_time / repeat);

    float our_time = 0.0;
    for (size_t i = 0; i < repeat; i++) {
        CUDA_CALL(hipEventRecord(start, stream));
        gemm_simple<half, 128, 128, 32>(c_our, a, b, m, n, k, stream);
        CUDA_CALL(hipEventRecord(end, stream));

        CUDA_CALL(hipEventSynchronize(end));
        float ms;
        CUDA_CALL(hipEventElapsedTime(&ms, start, end));
        our_time += ms;
    }
    printf("our time: %.3f ms\n", our_time / repeat);

    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(end));
    CUDA_CALL(hipStreamDestroy(stream));
}

int main(int argc, char **argv) {
    test(81920, 256, 256, 100);
    return EXIT_SUCCESS;
}
